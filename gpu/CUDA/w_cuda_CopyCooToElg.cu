#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "cintrf.h"
#include "vectordev.h"

#define THREAD_BLOCK 256
#define MIN(A,B) ( (A)<(B) ? (A) : (B) )
#define SQUARE(x) ((x)*(x))
#define GET_ADDR(a,ix,iy,nc) a[(nc)*(ix)+(iy)]
#define GET_VAL(a,ix,iy,nc) (GET_ADDR(a,ix,iy,nc))

#ifdef __cplusplus
extern "C" {
#endif


void w_cuda_CopyCooToElg(spgpuHandle_t handle, int nr, int nc, int nza, int hacksz, int ldv, int nzm,
		   int *rS,int *devIdisp, int *devJa, double *devVal,
		   int *rP, double *cM);


/* void _w_Cuda_coeff_upd_a_krn(int ifirst, int ii, int j, int nrws, int *idxs, int dim1,int dim2, */
/* 			  int *vi, int *vj, double *va, int *idxdiag, double *vdiag, */
/* 			  int nr, int nc, int *lidxs, double *vu,  */
/* 			     double c1, double c6); */

#ifdef __cplusplus
}
#endif



 

__global__ void _w_Cuda_cpy_coo_2_elg_krn(int ii, int nrws, int nr, int nza, int hacksz, int ldv, int nzm,
			  int *rS, int *devIdisp, int *devJa, double *devVal,  int *rP, double *cM)
{
  int ir, k, ipnt, rsz;
  int ki = threadIdx.x + blockIdx.x * (THREAD_BLOCK);
  int i=ii+ki; 

  if (ki >= nrws) return; 
  if (i >= nr) return; 

  ipnt=devIdisp[i];
  rsz=rS[i];
  ir = i;
  for (k=0; k<rsz; k++) {
    rP[ir] = devJa[ipnt];
    cM[ir] = devVal[ipnt];
    ir += ldv;
    ipnt++;
  }
  for (k=rsz; k<nzm; k++) {
    rP[ir] = i;
    cM[ir] = 0.0;
    ir += ldv;
  }
    
}    
  




void _w_Cuda_cpy_coo_2_elg(spgpuHandle_t handle, int nrws, int i, int nr, int nza, int hacksz, int ldv, int nzm,
			  int *rS,int *devIdisp, int *devJa, double *devVal,  int *rP, double *cM)
{
  dim3 block (THREAD_BLOCK, 1);
  dim3 grid ((nrws + THREAD_BLOCK - 1) / THREAD_BLOCK);

  _w_Cuda_cpy_coo_2_elg_krn 
    <<< grid, block, 0, handle->currentStream >>>(i,nrws, nr, nza, hacksz, ldv, nzm,
						  rS,devIdisp,devJa,devVal,rP,cM);

}




void w_cuda_CopyCooToElg(spgpuHandle_t handle, int nr, int nc, int nza, int hacksz, int ldv, int nzm,
		   int *rS,int *devIdisp, int *devJa, double *devVal,
		   int *rP, double *cM)
{ int i,j,k, nrws;
  //int maxNForACall = THREAD_BLOCK*handle->maxGridSizeX;
  int maxNForACall = max(handle->maxGridSizeX, THREAD_BLOCK*handle->maxGridSizeX);
  

  //fprintf(stderr,"Loop on j: %d\n",j); 
  for (i=0; i<nr; i+=nrws) {
    nrws = MIN(maxNForACall, nr - i);
    //fprintf(stderr,"ifirst: %d i : %d nrws: %d i + ifirst + (nrws -1) -1 %d \n",ifirst,i,nrws,i + ifirst + (nrws -1) -1);
    _w_Cuda_cpy_coo_2_elg(handle,nrws,i, nr, nza, hacksz, ldv, nzm,
			  rS,devIdisp, devJa, devVal,  rP, cM);
  }
}
